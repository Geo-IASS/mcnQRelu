#include "hip/hip_runtime.h"
// @file quickrelu_gpu.cu
// @brief quickrelu CUDA implementation 
// (this code is based on the implementation provided in caffe)
// @author Samuel Albanie
// @author Andrea Vedaldi

/*
Copyright (C) 2017 Samuel Albanie and Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "quickrelu.hpp"
#include <bits/data.hpp>
#include <assert.h>
#include <float.h>
#include <cstdio>

/* ------------------------------------------------------------ */
/*                                                      kernels */
/* ------------------------------------------------------------ */

template <typename T>
__global__ void reluForwardKernel(const int numThreads,
                                  const T* in, 
                                  T* out,
                                  T leak) 
{
    // Grid stride-loop 
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; 
             index < numThreads ; 
             index += blockDim.x * gridDim.x) 
    {
        out[index] = in[index] > 0 ? in[index] : in[index] * leak;
    }
}

template <typename T>
__global__ void reluBackwardKernel(const int numThreads,
                                   const T* in, 
                                   const T* der, 
                                   T* out,
                                   T leak) 
{
    // Grid stride-loop 
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; 
             index < numThreads ; 
             index += blockDim.x * gridDim.x) 
    {
        out[index] = in[index] > 0 ? der[index] : der[index] * leak;
    }
}

/* ------------------------------------------------------------ */
/*                                              kernel wrappers */
/* ------------------------------------------------------------ */

template <typename T>
void reluForwardGPU(const int numThreads,
                    const T* in, 
                    T* out,
                    T leak) 
{
    int numBlocks = (numThreads + 511) / 512 ;
    reluForwardKernel<T><<<numBlocks, 512>>>(numThreads, in, out, leak) ;
    hipError_t status = hipPeekAtLastError() ;
    if (status != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(status));
        exit(-1) ;
    }
}

template <typename T>
void reluBackwardGPU(const int numThreads,
                    const T* in, 
                    const T* der, 
                    T* out,
                    T leak) 
{
    int numBlocks = (numThreads + 511) / 512 ;
    reluBackwardKernel<T><<<numBlocks, 512>>>(numThreads, in, der, out, leak) ;
    hipError_t status = hipPeekAtLastError() ;
    if (status != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(status));
        exit(-1) ;
    }
}


namespace vl { namespace impl {

  template<typename T>
  struct quickrelu<vl::VLDT_GPU, T>
  {

    /* ------------------------------------------------------------ */
    /*                                                      forward */
    /* ------------------------------------------------------------ */
    static vl::ErrorCode
    forward(Context& context,
            T* output,
            T const* data,
            T const leak,
            size_t outSize)
    {
      reluForwardGPU<T>(outSize, data, output, leak) ;

      hipError_t status = hipPeekAtLastError() ;
      return (status == hipSuccess) ? vl::VLE_Success : vl::VLE_Cuda ;
    }


    /*------------------------------------------------------------- */
    /*                                                     backward */
    /* ------------------------------------------------------------ */

    static vl::ErrorCode
    backward(Context& context,
             T* derData,
             T const* data,
             T const* derOutput,
             T const leak,
             size_t outSize)
    {
      reluBackwardGPU<T>(outSize, data, derOutput, derData, leak) ;

      hipError_t status = hipPeekAtLastError() ;
      return (status == hipSuccess) ? vl::VLE_Success : vl::VLE_Cuda ;
    }
  } ;

} } // namespace vl::impl

template struct vl::impl::quickrelu<vl::VLDT_GPU, float> ;

#ifdef ENABLE_DOUBLE
template struct vl::impl::quickrelu<vl::VLDT_GPU, double> ;
#endif
